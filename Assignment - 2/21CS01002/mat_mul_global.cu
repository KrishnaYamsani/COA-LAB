#include<stdio.h>
#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
#include<time.h>

using namespace std;

#define rep(i,a,b) for(int i = a;i<b;i++)

__global__ void matMul(int* A, int* B, int* C,int m,int n, int p)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int temp_sum = 0;
    if((row<m)&&(col<n))
    {
        rep(k,0,p)
        {   
            temp_sum += A[row*p + k]*B[k*n + col];
        }
        C[row*n + col] = temp_sum;
    }
}

void init(int* A,int* B,int m,int n,int p)
{
    rep(i,0,m)
    {   
        rep(j,0,p)
            *(A+i*p+j) = 1;
    }

    rep(i,0,p)
    {   
        rep(j,0,n)
             *(B+i*n+j) = 2;
    }
   
}

int main()
{
    int *h_a, *h_b, *h_c; //host pointers
    int *d_a, *d_b, *d_c; //device pointers
    
    int m,p,n;

    m = 4, p = 2, n = 3; 

    size_t bytes_a = m*p*sizeof(int);
    size_t bytes_b = n*p*sizeof(int);
    size_t bytes_c = m*n*sizeof(int);

    h_a = (int*)malloc(bytes_a);
    h_b = (int*)malloc(bytes_b);
    h_c = (int*)malloc(bytes_c);

    init(h_a,h_b,m,n,p);

    hipMalloc(&d_a,bytes_a);
    hipMalloc(&d_b,bytes_b);
    hipMalloc(&d_c,bytes_c);

    int block_size = 16;
    int grid_size = (int)ceil((float)32/block_size);

    

    dim3 grid(grid_size,grid_size);
    dim3 threads(block_size,block_size);

    // cout<<block_size<<" "<<grid_size<<"\n";

    hipMemcpy(d_a,h_a,bytes_a,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,bytes_b,hipMemcpyHostToDevice);
    hipMemcpy(d_c,h_c,bytes_c,hipMemcpyHostToDevice);

    

    matMul<<<grid,threads>>> (d_a,d_b,d_c,m,n,p);

    hipMemcpy(h_c,d_c,bytes_c,hipMemcpyDeviceToHost);

    rep(i,0,m)
    {
        rep(j,0,n)
            cout<<*(h_c+i*n+j)<<" ";
        cout<<"\n";
    }
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}